
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include <stdio.h>
#include <string.h>
#include<time.h>
#define Size 10
#define patternSize 3
#define patternNum 20
#define ThreadNum 20 
#define BlockNum 1

__device__ void preKmp(char *x, int m, int kmpNext[])
{
	int i, j;
	i = 0;
	j = kmpNext[0] = -1;
	while(i < m)
	{
		while(j>-1 && x[i]!=x[j])
			j = kmpNext[j];
		i++;
		j++;
		if(x[i]==x[j])
			kmpNext[i] = kmpNext[j];
		else
			kmpNext[i] = j;

	}
}

__device__ void KMP(char *x, int m, char *y, int n,int *answer,int id)
{
	int i, j, kmpNext[Size];

	preKmp(x,m,kmpNext);
	i = j = 0;
	while(j < n)
	{
		while(i>-1 && x[i]!=y[j])
		{
		  	i = kmpNext[i];
		}
		i++;
		j++;
		if(i >= m)
		{
			i = kmpNext[i];
			answer[id]=j-1;	
		}

	}
}

__global__ void kmp_kernel(char *array,char *pattern,int *answer)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  char *p;
  p=&pattern[id*(patternSize+1)];
  KMP(p,patternSize,array,Size,answer,id);
   
}

int main(int argc,char *argv[])
{
  int i=0,j=0,tmp,*answer,*d_answer;
  hipError_t r;
  char *array,*b,*pattern;
  char *d_array,*d_pattern;



  srand(time(0));
  array=(char*)malloc(sizeof(char)*Size);
  b=(char*)malloc(sizeof(char)*26);
  pattern=(char*)malloc(sizeof(char)*(patternSize+1)*patternNum);
  answer=(int*)malloc(sizeof(int)*patternNum);
  /************************************
  *   cudaMalloc
  ************************************/

  hipMalloc((void**)&d_array,sizeof(char)*Size);
  hipMalloc((void**)&d_pattern,sizeof(char)*(patternSize+1)*patternNum);
  hipMalloc((void**)&d_answer,sizeof(int)*patternNum);


  b="abcdefghijklmnopqrstuvwxyz";
  for(i=0;i<Size;i++)
	array[i]=b[rand()%26];

  for(i=0;i<patternNum;i++)
  {
	tmp=rand()%(Size-patternSize);
	for(j=0;j<patternSize+1;j++)
	{
	  if(j!=patternSize)
	  {
		pattern[i*(patternSize+1)+j]=array[tmp++];
		printf("%d   %c\n",i,array[tmp-1]);
	  }
	  else
	  {
		printf("===================== %d   \n",j);
		pattern[i*(patternSize+1)+j]='\0';
		printf("%c\n",pattern[i*patternSize+j]);
	  }
	}
  }
  for(i=0;i<patternNum;i++)
  {
	answer[i]=0;
  }


  hipMemcpy(d_array,array,sizeof(char)*Size,hipMemcpyHostToDevice);
  
  hipMemcpy(d_pattern,pattern,sizeof(char)*(patternSize+1)*patternNum,hipMemcpyHostToDevice);
  
  hipMemcpy(d_answer,answer,sizeof(int)*patternNum,hipMemcpyHostToDevice);
  
  kmp_kernel<<<BlockNum, ThreadNum>>>(d_array, d_pattern, d_answer);

  hipMemcpy(answer, d_answer, sizeof(int)*patternNum, hipMemcpyDeviceToHost);


  printf("Array:\n");
  printf("Texto: %s\n", array);
  for(i=0;i<(patternSize+1)*patternNum;i++)
	  printf("%c", pattern[i]);
  printf("\n\n");
  for(i=0;i<patternNum;i++)
	printf("%d, %d\n", i, answer[i]);

  
  return 0;
}