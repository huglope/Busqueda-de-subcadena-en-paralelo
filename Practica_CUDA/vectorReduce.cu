#include "hip/hip_runtime.h"
/*
 * This example shows how to compute the reduction of the elements of a vector.
 *
 * Also, it shows how to measure the performance of block of threads of a kernel
 * accurately. Blocks are executed in parallel and out of order. Since there's no
 * synchronization mechanism between blocks, we measure the clock once for each block.
 *
 */

// Includes C
#include <stdio.h>
#include <assert.h>

// CUDA Runtime
#include <hip/hip_runtime.h>

// Includes Helper Functions
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

/*
 * vectorReduce
 *
 * This kernel computes a standard parallel reduction and evaluates the
 * time it takes to do that for each block. The timing results are stored in device memory.
 * 
 */
__global__ void time_and_reduce(float *vector_d, float *reduce_d, clock_t *times_d, int n)
{
	extern __shared__ float sdata[];
	
	// local thread ID (in block)
	int tidb = threadIdx.x;
	
    // global thread (ID in grid)
	int tidg = blockIdx.x * blockDim.x + tidb;
	
	// record the initial time for each block
	if (tidb == 0) {
		times_d[blockIdx.x] = clock();
	}
	
	// move data from global to shared memory
	sdata[tidb] = (tidg < n) ? vector_d[tidg] : 0;
	__syncthreads();
	
	// perform reduction in shared memory
	for (int s = blockDim.x/2; s > 0; s >>= 1) {
		if (tidb < s) {
			sdata[tidb] += sdata[tidb + s];
		}
		__syncthreads();
	}
	
	// write result for this block to global memory
	if (tidb == 0) {
		reduce_d[blockIdx.x] = sdata[0];
		times_d[blockIdx.x] = clock() - times_d[blockIdx.x];
	}
}

/*
 * Host main routine
 *
 */
int main(int argc, char **argv)
{
	// default parameter values
	int n = 1024, bsx = 32;
	
	// process command line arguments
	if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
		n = getCmdLineArgumentInt(argc, (const char **)argv, (const char *) "n");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "bsx")) {
		bsx = getCmdLineArgumentInt(argc, (const char **) argv, (const char *) "bsx");
	}
	size_t nBytes = n * sizeof(float);
	
	clock_t *clocks_h = NULL;
	clock_t *clocks_d = NULL;
	
	float elapsed_time = .0;
	float *vector_h, *reduce_h;	// host data
    float *vector_d, *reduce_d;	// device data
	
	// set the GPU to use
	int dev = 0;
	hipSetDevice(dev);
	
	// total number of thread blocks
	int nblocks = (n + bsx - 1) / bsx;

	// set kernel launch configuration
    dim3 grid(nblocks);
    dim3 block(bsx);
	
    // allocate host memory
    vector_h = (float *) malloc(nBytes);
    clocks_h = (clock_t *) malloc(nblocks * sizeof(clock_t));
    reduce_h = (float *) malloc(nblocks * sizeof(float));
	
	float acum = .0;
	// initialize host memory
    for(int i = 0; i < n; i++) {
        vector_h[i] = (float) 1;
		acum += 1.0;
	}
	
    // allocate device memory
    hipMalloc((void **) &vector_d, nBytes);
    hipMalloc((void **) &reduce_d, nblocks * sizeof(float));
	hipMalloc((void **) &clocks_d, nblocks * sizeof(clock_t));
	
	// create cuda events
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// insert stream 0 in start event
	hipEventRecord(start, 0);
	
    // copy data from host memory to device memory
    hipMemcpy(vector_d, vector_h, nBytes, hipMemcpyHostToDevice);
    
    // execute the kernel 
    printf("---> Running configuration: grid of %d blocks of %d threads (TOTAL: %d threads)\n", nblocks, bsx, nblocks * bsx);
    time_and_reduce<<<grid, block, bsx * sizeof(float)>>>(vector_d, reduce_d, clocks_d, n);

    // copy data from device memory to host memory
	hipMemcpy(reduce_h, reduce_d, nblocks * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(clocks_h, clocks_d, nblocks * sizeof(clock_t), hipMemcpyDeviceToHost);
	
	// insert stream 0 in stop event
	hipEventRecord(stop, 0);

    // using events to calculate the execution time        
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("---> Time spent executing by the GPU: %.2f\n", elapsed_time);
	
	long double avgElapsedClocks = 0;
    for (int i = 0; i < nblocks; i++) {
		avgElapsedClocks += (long double) clocks_h[i];
    }
    avgElapsedClocks = avgElapsedClocks / nblocks;
    printf("Average Clocks/Block = %Lf\n", avgElapsedClocks);

	// check the output for correctness
	float result = 0.0;
	for(int i = 0; i < nblocks; i++) { 
		result += reduce_h[i];
	}
	assert(result == (float) acum);
	
	// destroy events
	hipEventDestroy(start);
	hipEventDestroy(stop);

    // free host memory
    free(vector_h);
	free(reduce_h);
	free(clocks_h);
	
	// free device memory
    hipFree(vector_d);
    hipFree(reduce_d);
	hipFree(clocks_d);
	
    printf("\nTest PASSED\n");
	exit(EXIT_SUCCESS);
}
